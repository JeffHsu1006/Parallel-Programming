
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>

const int INF = ((1 << 30) - 1);
void input(char* inFileName);
void output(char* outFileName);

int ceil(int a, int b);
__global__ void phase1(int round, int n, int V, int* Dist, int B);
__global__ void phase2(int round, int n, int V, int* Dist, int B);
__global__ void phase3(int round, int n, int V, int* Dist, int B, int R, int rank);
extern __shared__ int S[];

int n, m, num_devices;
int *d_Dist_0, *d_Dist_1, *d_n;
int *Dist;


int main(int argc, char* argv[]) {

    input(argv[1]);
    int B = 32;
    int round = ceil(n, B);

    hipGetDeviceCount(&num_devices);
    int *d_Dist[num_devices];
    
    #pragma omp parallel num_threads(2)
    {
        int thread_id = omp_get_thread_num();
        hipSetDevice(thread_id);

        hipMalloc((void **)&d_Dist[thread_id], n * n * sizeof(int));
        hipMemcpy(d_Dist[thread_id], Dist, n * n * sizeof(int), hipMemcpyHostToDevice);
        
        hipMalloc((void **)&d_n, sizeof(int));
        hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

        dim3 grid1(1, 1);
        dim3 grid2(round, 2);
        dim3 grid3((round/2)+1, round);
        dim3 blk(B, B);

        for (int r = 0; r < round; ++r) {
            #pragma omp barrier
            if(n > B && r < (round/2) && thread_id == 1){
                hipMemcpyPeer((void*) &d_Dist[1][r * B * n], 1, (void*) &d_Dist[0][r * B * n], 0, B * n * sizeof(int));
  
            }else if(n > B && r >= (round/2) && thread_id == 0){
                if(r == (round-1))
                    hipMemcpyPeer((void*) &d_Dist[0][r * B * n], 0, (void*) &d_Dist[1][r * B * n], 1, (n - r * B) * n * sizeof(int));
                else
                    hipMemcpyPeer((void*) &d_Dist[0][r * B * n], 0, (void*) &d_Dist[1][r * B * n], 1, B * n * sizeof(int));
            }
            #pragma omp barrier
            phase1<<<grid1, blk, B*B*sizeof(int)>>>(r, n, n, d_Dist[thread_id], B);
            phase2<<<grid2, blk, 2*B*B*sizeof(int)>>>(r, n, n, d_Dist[thread_id], B);
            phase3<<<grid3, blk, 2*B*B*sizeof(int)>>>(r, n, n, d_Dist[thread_id], B, round, thread_id);
        }

        if(thread_id == 0)
            hipMemcpy(Dist, d_Dist[0], (round/2) * B * n * sizeof(int), hipMemcpyDeviceToHost);
        else if(n > B && thread_id == 1)
            hipMemcpy(&Dist[(round/2) * B * n], &d_Dist[1][(round/2) * B * n], (n - (round/2) * B) * n * sizeof(int), hipMemcpyDeviceToHost);

    }

    output(argv[2]);

    return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    Dist = (int*)malloc(n*n*sizeof(int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i * n + j] >= INF) Dist[i * n + j] = INF;
        }
    }
    fwrite(Dist, sizeof(int), n*n, outfile);
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void phase1(int round, int n, int V, int* Dist, int B){
    int s_i = threadIdx.y;
    int s_j = threadIdx.x;
    int i = round * B + s_i;
    int j = round * B + s_j;
    
    if((i < n && j < n))
        S[s_i * B + s_j] = Dist[i * V + j];
    __syncthreads();

    int tt = round * B;
    int ss = s_i * B;
    #pragma unroll
    for (int k = 0; k < B && tt + k < n; ++k) {
        if (S[ss + k] + S[k * B + s_j] < S[ss + s_j])
            S[ss + s_j] = S[ss + k] + S[k * B + s_j];
        
        __syncthreads();
    }
    if (i < n && j < n) Dist[i * V + j] = S[ss + s_j];
    __syncthreads();

}

__global__ void phase2(int round, int n, int V, int* Dist, int B){

    if (blockIdx.x == round) return;

    int* S_pivot = &S[0];
    int* S_dist = &S[B * B];

    int s_i = threadIdx.y;
    int s_j = threadIdx.x;
    int i = round * B + s_i;
    int j = round * B + s_j;
    
    int ss = s_i * B;

    if((i < n && j < n))
        S_pivot[ss + s_j] = Dist[i * V + j];
    __syncthreads();

    if (blockIdx.y == 0)
        j = blockIdx.x * B + s_j;
    else
        i = blockIdx.x * B + s_i;

    if (i >= n || j >= n) return;

    if((i < n && j < n))
        S_dist[ss + s_j] = Dist[i * V + j];
    __syncthreads();

    int tt = round * B;
    if(blockIdx.y == 1){
        #pragma unroll
        for (int k = 0; k < B && tt + k < n; ++k) {
            if (S_dist[ss + k] + S_pivot[k * B + s_j] < S_dist[ss + s_j])
                S_dist[ss + s_j] = S_dist[ss + k] + S_pivot[k * B + s_j];
        }
    }else{
        #pragma unroll
        for (int k = 0; k < B && tt + k < n; ++k) {
            if (S_pivot[ss + k] + S_dist[k * B + s_j] < S_dist[ss + s_j])
                S_dist[ss + s_j] = S_pivot[ss + k] + S_dist[k * B + s_j];
        }
    }
    
    if (i < n && j < n) Dist[i * V + j] = S_dist[ss + s_j];
    __syncthreads();
}

__global__ void phase3(int round, int n, int V, int* Dist, int B, int R, int rank){

    int block_i = blockIdx.x;
    int block_j = blockIdx.y;

    if(rank == 1)
        block_i += (R/2);

    if (block_i == round || block_j == round) return;

    int* S_pivot_row = &S[0];
    int* S_pivot_col= &S[B * B];

    int s_i = threadIdx.y;
    int s_j = threadIdx.x;
    int i = block_i * B + s_i;
    int j = block_j * B + s_j;
    int b_i = round * B + s_i;
    int b_j = round * B + s_j;

    int ss = s_i * B;
    
    if(i < n && b_j < n) S_pivot_row[ss + s_j] = Dist[i * V + b_j];
    if(j < n && b_i < n) S_pivot_col[ss + s_j] = Dist[b_i * V + j];
    __syncthreads();

    if (i >= n || j >= n) return;

    int dst = Dist[i * V + j];

    int tt = round * B;
    #pragma unroll
    for (int k = 0; k < B && tt + k < n; ++k) {
        int tmp_result = S_pivot_row[ss + k] + S_pivot_col[k * B + s_j];
        if (tmp_result < dst) dst = tmp_result;
    }
    
    Dist[i * V + j] = dst;
}
